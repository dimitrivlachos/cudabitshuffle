#include "hip/hip_runtime.h"
/*
 * CUDA Kernel for bit shuffling
 */

#include "cudabitshuffle.hpp"

//
inline auto cuda_error_string(hipError_t err) {
  const char *err_name = hipGetErrorName(err);
  const char *err_str = hipGetErrorString(err);
  printf("CUDA Error: %s: %s\n", err_name, err_str);
}
/// Raise an exception IF CUDA is in an error state, with the name and
/// description
inline auto cuda_throw_error() -> void {
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA Error: %s: %s\n", hipGetErrorName(err),
           hipGetErrorString(err));
  }
}

__global__ void cuda_bitshuffle(unsigned int *d_input, unsigned int *d_output,
                                int numElements, int numBits) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < numElements) {
    unsigned int input = d_input[idx];
    unsigned int output = 0;
    for (int i = 0; i < numBits; i++) {
      output |= ((input >> i) & 1) << (numBits - 1 - i);
    }
    d_output[idx] = output;
  }
}

__global__ void print_array_kernel(uint8_t *d_buffer, int length, int index) {
  int limit = min(index + 50, length);
  for (int i = index; i < limit; i++) {
    printf("%d ", d_buffer[i]);
  }
  printf("\n");
}

__global__ void test() { printf("Hello from CUDA\n"); }

// void nvc_compress(uint8_t *d_buffer) {
//   hipStream_t stream;
//   hipStreamCreate(&stream);

//   // Inirialize data on host
//   size_t *host_uncompressed_bytes;
//   const size_t chunk_size = 8192;
//   const batch_size =
// }

/**
 * @brief: Return the byteswapped value of a 64-bit header
 * @param ptr: Pointer to the 64-bit header
 * @return: The byteswapped value of the header
 */
uint64_t byteswap64(const void *ptr) {
  uint64_t value;
  memcpy(&value, ptr, sizeof(uint64_t));
  uint8_t *bytes = (uint8_t *)&value;
  uint8_t tmp;
  tmp = bytes[0];
  bytes[0] = bytes[7];
  bytes[7] = tmp;
  tmp = bytes[1];
  bytes[1] = bytes[6];
  bytes[6] = tmp;
  tmp = bytes[2];
  bytes[2] = bytes[5];
  bytes[5] = tmp;
  tmp = bytes[3];
  bytes[3] = bytes[4];
  bytes[4] = tmp;
  memcpy(&value, bytes, sizeof(uint64_t));
  return value;
}

/**
 * @brief: Return the byteswapped value of a 32-bit header
 * @param ptr: Pointer to the 32-bit header
 * @return: The byteswapped value of the header
 */
uint32_t byteswap32(void *ptr) {
  uint32_t value;
  memcpy(&value, ptr, sizeof(uint32_t));
  uint8_t *bytes = (uint8_t *)&value;
  uint8_t tmp;
  tmp = bytes[0];
  bytes[0] = bytes[3];
  bytes[3] = tmp;
  tmp = bytes[1];
  bytes[1] = bytes[2];
  bytes[2] = tmp;
  memcpy(&value, bytes, sizeof(uint32_t));
  return value;
}

void decompress_lz4_gpu(const uint8_t *compressed_data, size_t compressed_size,
                        uint8_t *decompressed_data, size_t decompressed_size) {
  hipStream_t stream;
  hipStreamCreate(&stream);

  const size_t chunk_size = 8192;
  const size_t batch_size = (compressed_size + chunk_size - 1) / chunk_size;

  // Allocate device memory for compressed data
  uint8_t *device_compressed_data;
  hipMalloc(&device_compressed_data, compressed_size);
  hipMemcpyAsync(device_compressed_data, compressed_data, compressed_size,
                  hipMemcpyHostToDevice, stream);

  // Allocate device memory for uncompressed data
  uint8_t *device_decompressed_data;
  hipMalloc(&device_decompressed_data, decompressed_size);

  // Allocate temporary buffer for decompression
  size_t decomp_temp_bytes;
  nvcompBatchedLZ4DecompressGetTempSize(batch_size, chunk_size,
                                        &decomp_temp_bytes);
  void *device_decomp_temp;
  hipMalloc(&device_decomp_temp, decomp_temp_bytes);

  // Allocate space for compressed chunk sizes
  size_t *device_compressed_bytes;
  hipMalloc(&device_compressed_bytes, sizeof(size_t) * batch_size);

  // Allocate space for uncompressed chunk sizes
  size_t *device_decompressed_bytes;
  hipMalloc(&device_decompressed_bytes, sizeof(size_t) * batch_size);

  // Decompress the data
  nvcompStatus_t decomp_res = nvcompBatchedLZ4DecompressAsync(
      reinterpret_cast<const void *const *>(&device_compressed_data),
      &compressed_size, device_decompressed_bytes, device_decompressed_bytes,
      batch_size, device_decomp_temp, decomp_temp_bytes,
      reinterpret_cast<void *const *>(&device_compressed_data), nullptr,
      stream);

  if (decomp_res != nvcompSuccess) {
    std::cerr << "Failed decompression!" << std::endl;
    assert(decomp_res == nvcompSuccess);
  }

  // Copy the decompressed data back to host memory
  hipMemcpyAsync(decompressed_data, device_decompressed_data,
                  decompressed_size, hipMemcpyDeviceToHost, stream);

  hipStreamSynchronize(stream);

  // Free device memory
  hipFree(device_compressed_data);
  hipFree(device_decompressed_data);
  hipFree(device_decomp_temp);
  hipFree(device_compressed_bytes);
  hipFree(device_decompressed_bytes);
  hipStreamDestroy(stream);
}

void run_test() {
  test<<<1, 1>>>();
  cuda_throw_error();
  hipDeviceSynchronize();
}

void print_array(uint8_t *d_buffer, int length, int index) {
  print_array_kernel<<<1, 1>>>(d_buffer, length, index);
  cuda_throw_error();
  hipDeviceSynchronize();
}