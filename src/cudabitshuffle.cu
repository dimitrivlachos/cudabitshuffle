#include "hip/hip_runtime.h"
/*
 * CUDA Kernel for bit shuffling
 */

#include "cudabitshuffle.hpp"

#define CHUNK_SIZE 8192

// Define the pixel type
using pixel_t = H5Read::image_type;

inline auto cuda_error_string(hipError_t err) {
  const char *err_name = hipGetErrorName(err);
  const char *err_str = hipGetErrorString(err);
  printf("CUDA Error: %s: %s\n", err_name, err_str);
}
/// Raise an exception IF CUDA is in an error state, with the name and
/// description
inline auto cuda_throw_error() -> void {
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA Error: %s: %s\n", hipGetErrorName(err),
           hipGetErrorString(err));
  } else {
    printf("No CUDA error\n");
  }
}

// __global__ void cuda_bitshuffle(unsigned int *d_input, unsigned int
// *d_output,
//                                 int numElements, int numBits) {
//   int idx = blockIdx.x * blockDim.x + threadIdx.x;
//   if (idx < numElements) {
//     unsigned int input = d_input[idx];
//     unsigned int output = 0;
//     for (int i = 0; i < numBits; i++) {
//       output |= ((input >> i) & 1) << (numBits - 1 - i);
//     }
//     d_output[idx] = output;
//   }
// }

/**
 * @brief: Swap the bytes of a 64-bit integer in place
 * @param: ptr - pointer to the 64-bit integer
 */
void byteswap64(void *ptr) {
  uint8_t *bytes = (uint8_t *)ptr;
  uint8_t tmp;
  tmp = bytes[0];
  bytes[0] = bytes[7];
  bytes[7] = tmp;
  tmp = bytes[1];
  bytes[1] = bytes[6];
  bytes[6] = tmp;
  tmp = bytes[2];
  bytes[2] = bytes[5];
  bytes[5] = tmp;
  tmp = bytes[3];
  bytes[3] = bytes[4];
  bytes[4] = tmp;
}

/**
 * @brief: Swap the bytes of a 32-bit integer in place
 * @param: ptr - pointer to the 32-bit intege
 */
void byteswap32(void *ptr) {
  uint8_t *bytes = (uint8_t *)ptr;
  uint8_t tmp;
  tmp = bytes[0];
  bytes[0] = bytes[3];
  bytes[3] = tmp;
  tmp = bytes[1];
  bytes[1] = bytes[2];
  bytes[2] = tmp;
}

template <typename T>
__global__ void print_array_kernel(T *d_array, size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    printf("%d: %f\n", idx, static_cast<float>(d_array[idx]));
  }
}

template <> __global__ void print_array_kernel<int>(int *d_array, size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    printf("%d: %d\n", idx, d_array[idx]);
  }
}

template <>
__global__ void print_array_kernel<double>(double *d_array, size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    printf("%d: %lf\n", idx, d_array[idx]);
  }
}

template <typename T> void print_array(T *d_array, size_t size) {
  // Define block size and grid size
  int blockSize = 256;
  int gridSize = (size + blockSize - 1) / blockSize;

  // Launch kernel to print the array
  print_array_kernel<<<gridSize, blockSize>>>(d_array, size);

  // Synchronize device to ensure all prints are done
  cuda_throw_error();
  hipDeviceSynchronize();
}

/**
 * @brief Get the absolute block offsets from the compressed chunk data
 * as well as the size of each compressed chunk. We use absolute offsets
 * so that we can easily calculate the pointer to each block on the GPU
 * @param h_buffer The compressed data
 * @param block_offsets The calculated absolute block offsets
 * @param block_sizes The calculated sizes of each block
 * @param batch_size The number of blocks
 */
void get_block_size_and_offset(uint8_t *h_buffer, size_t *block_offsets,
                               size_t *block_sizes, size_t batch_size) {
  // Byteswap the header
  byteswap64(h_buffer);
  byteswap32(h_buffer + 8);

  // Now byte swap the block headers
  uint8_t *block = h_buffer + 12; // Skip header
  printf("Block: %p\n", block);
  // print first 24 bytes of the block
  for (int i = 0; i < 24; i++) {
    printf("%d ", block[i]);
  }
  printf("\n");

  uint32_t image_size = (uint32_t) * (uint64_t *)h_buffer; // Get the image size
  uint32_t n_block = image_size / CHUNK_SIZE; // Calculate the number of blocks
  if (image_size % CHUNK_SIZE) { // If there is a remainder, add one more block
    n_block++;
  }

  // Ensure we do not exceed the allocated batch_size
  if (n_block > batch_size) {
    printf("Error: Number of blocks exceeds batch size.\n");
    return;
  }

  int cumulative_offset = 4;            // The cumulative offset of the blocks
  block_offsets[0] = cumulative_offset; // The first block starts at 0

  for (int i = 0; i < n_block; i++) {   // Iterate over the blocks
    byteswap32(block);                  // Byteswap the block header
    uint32_t next = *(uint32_t *)block; // Get the size of the block

    block_sizes[i] = next; // Add the size of the block to the block sizes
    cumulative_offset += next + 4; // Accumulate the offset
    block_offsets[i + 1] =
        cumulative_offset; // Add the offset to the block offsets
    block += next + 4;     // Move to the next block

    if (i < 10 || i > 4400) {
      printf("%d: Next: %d, Cumulative: %d, Block: %p, Block offset size: %d\n",
             i, next, cumulative_offset, block, i + 2);
    }
  }

  // Print the sizes of block_offsets and block_sizes
  printf("Block offsets size: %zu\n", n_block + 1);
  printf("Block sizes size: %zu\n", n_block);
}

/**
 * @brief Kernel to convert block offsets to gpu pointers, this replaces
 * the need for a for loop and more optimally utilises the GPU
 * @param d_data The compressed data on the device
 * @param d_block_offsets The offsets of the compressed blocks
 * @param d_ptr_list The output pointers to the compressed blocks
 * @param batch_size The number of blocks
 */
__global__ void block_offset_to_pointers_kernel(const uint8_t *d_data,
                                                const size_t *d_block_offsets,
                                                void **d_ptr_list,
                                                int batch_size) {
  // Get the thread index
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < batch_size) {
    // Initialize the pointer to the beginning of the compressed data
    const uint8_t *current_ptr = d_data;

    // Move the pointer to the block offset
    current_ptr += d_block_offsets[i];

    if (i < 10 || i > 4400) {
      printf("Current pointer %d: %p   ", i, current_ptr);
      // // print first 24 bytes of the block
      // for (int i = 0; i < 24; i++) {
      //   printf("%d ", current_ptr[i]);
      // }
      // printf("\n");
    }

    // Set the pointer to the compressed block
    d_ptr_list[i] = (void *)current_ptr;
  }
}

/**
 * @brief Converts the block offsets to gpu memory pointers
 * @param d_data The compressed data on the device
 * @param block_offsets The offsets of the compressed blocks
 * @param d_compressed_ptrs The output pointers to the compressed blocks
 */
void block_offset_to_pointers(const uint8_t *d_data, size_t *d_block_offsets,
                              int batch_size, void **d_compressed_ptrs) {
  // Launch kernel
  dim3 blocks((batch_size + 255) /
              256); // Create enough blocks to cover all the indices
  dim3 threads(256);
  block_offset_to_pointers_kernel<<<blocks, threads>>>(
      d_data, d_block_offsets, d_compressed_ptrs, batch_size);
  cuda_throw_error();
  hipDeviceSynchronize();
  // Free device memory
  hipFree(d_block_offsets);
  printf("\n");
}

__global__ void print_pointers_kernel(void **d_uncompressed_ptrs,
                                      size_t batch_size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < batch_size) {
    printf("Pointer %d: %p\n", idx, d_uncompressed_ptrs[idx]);
  }
}

__global__ void prefix_sum_kernel(size_t *d_uncompressed_bytes,
                                  size_t *d_prefix_sum_bytes,
                                  size_t *batch_size) {
  // Calculate the global thread ID
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= *batch_size) {
    return;
  }

  // Initialize the prefix sum
  size_t prefix_sum = 0;

  // Iterate over the uncompressed sizes
  for (size_t i = 0; i < idx; ++i) {
    // Add the current size to the prefix sum
    prefix_sum += d_uncompressed_bytes[i];
  }

  // Set the prefix sum for the current chunk
  d_prefix_sum_bytes[idx] = prefix_sum;
}

/**
 * @brief Decompresses the data using bitshuffle and LZ4 on the GPU
 */
void bshuf_decompress_lz4_gpu(uint8_t *h_compressed_data,
                              const size_t image_size, uint8_t *out) {
  int image_size_bytes = image_size * sizeof(pixel_t);

  // Print the first 24 bytes of the compressed data
  for (int i = 0; i < 24; i++) {
    printf("%d ", h_compressed_data[i]);
  }
  printf("\n");

  // Allocate device memory for the compressed data and copy from host to device
  uint8_t *d_compressed_data;
  hipMalloc(&d_compressed_data, image_size_bytes);
  hipMemcpy(d_compressed_data, h_compressed_data + 12, image_size_bytes,
             hipMemcpyHostToDevice);
  printf("d_buffer size: %zu\n", image_size_bytes);

  // Set up CUDA stream for asynchronous operations
  hipStream_t stream;
  hipStreamCreate(&stream);

  // Determine the number of blocks (batch size)
  size_t chunk_size = CHUNK_SIZE;
  size_t batch_size = (image_size_bytes + chunk_size - 1) / chunk_size + 1;

  printf("Batch size: %zu\n", batch_size);

  // Calculate block offsets and sizes from compressed data
  size_t *managed_block_offsets;
  size_t *managed_compressed_bytes;

  hipMallocManaged(&managed_block_offsets, batch_size * sizeof(size_t));
  hipMallocManaged(&managed_compressed_bytes, batch_size * sizeof(size_t));

  get_block_size_and_offset(h_compressed_data, managed_block_offsets,
                            managed_compressed_bytes, batch_size);

  // Allocate device memory for pointers to compressed and uncompressed data
  void **d_compressed_ptrs;
  // size_t *d_compressed_bytes;
  size_t *d_uncompressed_bytes;
  void **d_uncompressed_ptrs;
  hipMalloc(&d_compressed_ptrs, sizeof(uint8_t *) * batch_size);
  // hipMalloc(&d_compressed_bytes, sizeof(size_t) * batch_size);
  hipMalloc(&d_uncompressed_bytes, sizeof(size_t) * batch_size);
  hipMalloc(&d_uncompressed_ptrs, sizeof(size_t) * batch_size);

  // Print the first 10 block sizes
  for (int i = 0; i < 10; i++) {
    printf("Block size %d: %d\n", i, managed_compressed_bytes[i]);
  }

  print_array(managed_compressed_bytes, 10);

  // Convert block offsets to pointers on the GPU
  block_offset_to_pointers(d_compressed_data, managed_block_offsets, batch_size,
                           d_compressed_ptrs);

  // Decompression size and temporary buffer setup
  size_t decomp_temp_bytes;
  nvcompBatchedLZ4DecompressGetTempSize(batch_size, chunk_size,
                                        &decomp_temp_bytes);
  printf("Decompression temp size: %zu\n", decomp_temp_bytes);
  void *d_decomp_temp;
  hipMalloc(&d_decomp_temp, decomp_temp_bytes);

  // Setup for decompression error handling
  nvcompStatus_t *device_statuses;
  hipMalloc(&device_statuses, sizeof(nvcompStatus_t) * batch_size);

  printf("Getting decompressed size\n");
  // Get the size of the decompressed data asynchronously
  nvcompBatchedLZ4GetDecompressSizeAsync(
      d_compressed_ptrs, managed_compressed_bytes, d_uncompressed_bytes,
      batch_size, stream);

  hipStreamSynchronize(stream);

  printf("Uncompressed sizes\n");
  print_array(d_uncompressed_bytes, 10);
  print_array(d_uncompressed_bytes + batch_size - 10, 10);
  printf("\n");

  // Calculate the uncompressed pointers from the sizes
  size_t *d_batch_size;
  size_t *d_prefix_sum_bytes; // List of absolute offsets
  hipMalloc(&d_prefix_sum_bytes, batch_size * sizeof(size_t));
  hipMalloc(&d_batch_size, sizeof(size_t));
  hipMemcpy(d_batch_size, &batch_size, sizeof(size_t), hipMemcpyHostToDevice);
  dim3 blocks((batch_size + 255) / 256);
  dim3 threads(256);
  prefix_sum_kernel<<<blocks, threads>>>(d_uncompressed_bytes,
                                         d_prefix_sum_bytes, d_batch_size);
  cuda_throw_error();
  hipDeviceSynchronize();

  block_offset_to_pointers((uint8_t *)d_uncompressed_ptrs, d_prefix_sum_bytes,
                           batch_size, d_uncompressed_ptrs);

  // Perform the decompression
  printf("Decompressing\n");
  nvcompStatus_t decomp_res = nvcompBatchedLZ4DecompressAsync(
      d_compressed_ptrs, managed_compressed_bytes, d_uncompressed_bytes,
      nullptr, batch_size, d_decomp_temp, decomp_temp_bytes,
      d_uncompressed_ptrs, device_statuses, stream);

  // Check results of the decompression
  if (decomp_res != nvcompSuccess) {
    printf("Error in decompression\n");
  } else {
    printf("Decompression successful\n");
  }

  // Synchronize stream to ensure all operations are complete
  hipStreamSynchronize(stream);

  // Check decompression status for each block
  nvcompStatus_t *host_statuses = new nvcompStatus_t[batch_size];
  hipMemcpy(host_statuses, device_statuses,
             batch_size * sizeof(nvcompStatus_t), hipMemcpyDeviceToHost);
  for (int i = 0; i < batch_size; ++i) {
    if (host_statuses[i] != nvcompSuccess) {
      printf("Decompression error on block %d: %d\n", i, host_statuses[i]);
    }
  }

  hipStreamSynchronize(stream);

  print_array(d_uncompressed_bytes, 10);

  // Cleanup
  // delete[] host_statuses;
  hipFree(d_compressed_data);
  hipFree(d_compressed_ptrs);
  // hipFree(d_compressed_bytes);
  hipFree(d_uncompressed_bytes);
  hipFree(d_uncompressed_ptrs);
  hipFree(d_decomp_temp);
  hipFree(device_statuses);
  hipStreamDestroy(stream);
}