#include "hip/hip_runtime.h"
/*
 * CUDA Kernel for bit shuffling
 */

#include "cudabitshuffle.hpp"
#include <iostream>
#include <stdio.h>

#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      printf("CUDA Error: %s: %s\n", hipGetErrorName(err),                    \
             hipGetErrorString(err));                                         \
    }                                                                          \
  } while (0)

//
inline auto cuda_error_string(hipError_t err) {
  const char *err_name = hipGetErrorName(err);
  const char *err_str = hipGetErrorString(err);
  printf("CUDA Error: %s: %s\n", err_name, err_str);
}
/// Raise an exception IF CUDA is in an error state, with the name and
/// description
inline auto cuda_throw_error() -> void {
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA Error: %s: %s\n", hipGetErrorName(err),
           hipGetErrorString(err));
  }
}

__global__ void cuda_bitshuffle(unsigned int *d_input, unsigned int *d_output,
                                int numElements, int numBits) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < numElements) {
    unsigned int input = d_input[idx];
    unsigned int output = 0;
    for (int i = 0; i < numBits; i++) {
      output |= ((input >> i) & 1) << (numBits - 1 - i);
    }
    d_output[idx] = output;
  }
}

__global__ void print_array_kernel(uint8_t *d_buffer, int length, int index) {
  int limit = min(index + 50, length);
  for (int i = index; i < limit; i++) {
    printf("%d ", d_buffer[i]);
  }
  printf("\n");
}

__global__ void test() { printf("Hello from CUDA\n"); }

void nvc_decompress(uint8_t *d_buffer) {
  using namespace nvcomp;
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  uint8_t *comp_buffer = d_buffer;

  auto decomp_nvcomp_manager = create_manager(comp_buffer, stream);

  DecompressionConfig decomp_config =
      decomp_nvcomp_manager->configure_decompression(comp_buffer);
  uint8_t *res_decomp_buffer;
  CUDA_CHECK(hipMalloc(&res_decomp_buffer, decomp_config.decomp_data_size));

  decomp_nvcomp_manager->decompress(res_decomp_buffer, comp_buffer,
                                    decomp_config);
  print_array(res_decomp_buffer, decomp_config.decomp_data_size, 0);
}

void run_test() {
  test<<<1, 1>>>();
  cuda_throw_error();
  hipDeviceSynchronize();
}

void print_array(uint8_t *d_buffer, int length, int index) {
  print_array_kernel<<<1, 1>>>(d_buffer, length, index);
  cuda_throw_error();
  hipDeviceSynchronize();
}